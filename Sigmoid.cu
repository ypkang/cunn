#include "hip/hip_runtime.h"
#include "utils.h"
#include "THCApply.cuh"

struct sigmoidupdateOutput_functor
{
  __device__ void operator()(float* output, const float* input) const
  {
    *output = 1./(1.+ exp(-*input));
  }
};

static int cunn_Sigmoid_updateOutput(lua_State *L)
{
  //double ts = get_ts();
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THAssert(THCudaTensor_checkGPU(state, 2, input, output));
  THCudaTensor_resizeAs(state, output, input);
  //double pre_tensor = get_ts() - ts;
  double ts = get_ts();
  THCudaTensor_pointwiseApply2(state, output, input, sigmoidupdateOutput_functor());
  //double tensor = get_ts() - ts;
  //double post_tensor = 0.0;
  double pointwiseApply2_sigmoid = get_ts() - ts;
  //std::cout<<"Sigmoid__pre_tensor|"<<pre_tensor<<std::endl;
  //std::cout<<"Sigmoid__tensor|"<<tensor<<std::endl;
  //std::cout<<"Sigmoid__post_tensor|"<<post_tensor<<std::endl;
  std::cout<<std::fixed<<"pointwiseApply2_sigmoid,"<<pointwiseApply2_sigmoid<<std::endl;
  return 1;
}

struct sigmoidupdateGradInput_functor
{
  __device__ void operator()(float* gradInput, const float* output, const float* gradOutput) const
  {
    *gradInput = *gradOutput * (1.-*output) * *output;
  }
};

static int cunn_Sigmoid_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  THAssert(THCudaTensor_checkGPU(state, 3, output, gradOutput, gradInput));
  THCudaTensor_resizeAs(state, gradInput, output);
  THCudaTensor_pointwiseApply3(state, gradInput, output, gradOutput, sigmoidupdateGradInput_functor());
  return 1;
}

static const struct luaL_Reg cunn_Sigmoid__ [] = {
  {"Sigmoid_updateOutput", cunn_Sigmoid_updateOutput},
  {"Sigmoid_updateGradInput", cunn_Sigmoid_updateGradInput},
  {NULL, NULL}
};

void cunn_Sigmoid_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_Sigmoid__, "nn");
  lua_pop(L,1);
}
