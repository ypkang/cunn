#include "hip/hip_runtime.h"
#include "utils.h"
#include "THCApply.cuh"

struct tanhupdateOutput_functor
{
  __device__ void operator()(float* output, const float* input) const
  {
    *output = tanh(*input);
  }
};

static int cunn_Tanh_updateOutput(lua_State *L)
{
  //double ts = get_ts();
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THAssert(THCudaTensor_checkGPU(state, 2, input, output));
  THCudaTensor_resizeAs(state, output, input);
  //double pre_tensor = get_ts() - ts;
  double ts = get_ts();
  THCudaTensor_pointwiseApply2(state, output, input, tanhupdateOutput_functor());
  double pointwiseApply2_tanh = get_ts() - ts;
  //double tensor = get_ts() - ts;
  //double post_tensor = 0.0;
  //std::cout<<"Tanh__pre_tensor|"<<pre_tensor<<std::endl;
  //std::cout<<"Tanh__tensor|"<<tensor<<std::endl;
  //std::cout<<"Tanh__post_tensor|"<<post_tensor<<std::endl;
  std::cout<<std::fixed<<"pointwiseApply2_tanh,"<<pointwiseApply2_tanh<<std::endl;
  return 1;
}

struct tanhupdateGradInput_functor
{
  __device__ void operator()(float* gradInput, const float* output, const float* gradOutput) const
  {
    *gradInput = *gradOutput * (1 - *output * *output);
  }
};

static int cunn_Tanh_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  THAssert(THCudaTensor_checkGPU(state, 3, output, gradOutput, gradInput));
  THCudaTensor_resizeAs(state, gradInput, output);
  THCudaTensor_pointwiseApply3(state, gradInput, output, gradOutput, tanhupdateGradInput_functor());
  return 1;
}

static const struct luaL_Reg cunn_Tanh__ [] = {
  {"Tanh_updateOutput", cunn_Tanh_updateOutput},
  {"Tanh_updateGradInput", cunn_Tanh_updateGradInput},
  {NULL, NULL}
};

void cunn_Tanh_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_Tanh__, "nn");
  lua_pop(L,1);
}
